#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "fp.h"
#include <stdint.h>

uint64_t clock_now_cuda(void) {
  unsigned int tbl, tbu0, tbu1;

  do {
    __asm__ __volatile__("mftbu %0" : "=r"(tbu0));
    __asm__ __volatile__("mftb %0" : "=r"(tbl));
    __asm__ __volatile__("mftbu %0" : "=r"(tbu1));
  } while (tbu0 != tbu1);
  return (((uint64_t)tbu0) << 32) | tbl;
}

f3 inline __device__ sub(f3 a, f3 b) {
  f3 out;
  out.data[0] = a.data[0]-b.data[0];
  out.data[1] = a.data[1]-b.data[1];
  out.data[2] = a.data[2]-b.data[2];
  return out;
}
double inline __device__ dot(f3 a, f3 b) {
  double out = 0;
  out += a.data[0]*b.data[0];
  out += a.data[1]*b.data[1];
  out += a.data[2]*b.data[2];
  return out;
}
double inline __device__ len(f3 a) {
  double len = 0;
  len += a.data[0]*a.data[0];
  len += a.data[1]*a.data[1];
  len += a.data[2]*a.data[2];
  return sqrt(len);
}



void __global__ calculate(FP** lines, double* out) {
  extern __shared__ double lineFactor[];
  FP p = lines[blockIdx.x][threadIdx.x];
  f3 iToJ = sub(p.a,p.b);
  f3 jToI = sub(p.b,p.a);
  double iCos = dot(p.anorm, iToJ) / len(iToJ);
  double jCos = dot(p.bnorm, jToI) / len(jToI);
  lineFactor[threadIdx.x] = fabs(iCos*jCos)/ (len(iToJ)*len(jToI));
  __syncthreads();
  for(unsigned int s=blockDim.x/2; s > 0; s >>= 1) {
    if(threadIdx.x < s) {
      lineFactor[threadIdx.x] += lineFactor[threadIdx.x+s];
    }
    __syncthreads();
  }
  if(threadIdx.x == 0) {
    out[blockIdx.x] = lineFactor[0];
  }
}

double* getFormFactor(FP** lines, uint64_t* memoryTime, uint64_t* cudaTime, int count, int samples, int rank) {
  int c;
  hipGetDeviceCount(&c);
  hipSetDevice(rank % c);
  double* cudaOut;
  FP** cudaLines;
  uint64_t memoryStart = clock_now_cuda();
  hipMallocManaged(&cudaLines, count*sizeof(FP*));
  for(int i = 0; i < count; i++) {
    hipMallocManaged(cudaLines+i,samples*sizeof(FP));
    for(int j = 0; j < samples; j++) {
      cudaLines[i][j] = lines[i][j];
    }
  }
  hipMallocManaged(&cudaOut, count*sizeof(double));
  uint64_t memoryEnd = clock_now_cuda();
  *memoryTime += (memoryEnd-memoryStart);
  uint64_t computeStart = clock_now_cuda();
  calculate<<<count, samples, samples*sizeof(double)>>>(cudaLines, cudaOut);
  hipDeviceSynchronize();
  uint64_t computeEnd = clock_now_cuda();
  *cudaTime += (computeEnd-computeStart);
  memoryStart = clock_now_cuda();
  double* out = (double*)calloc(count, sizeof(double));
  for(int i = 0; i < count; i++) {
    out[i] = cudaOut[i];
    hipFree(cudaLines[i]);
  }
  hipFree(cudaLines);
  hipFree(cudaOut);
  memoryEnd = clock_now_cuda();
  *memoryTime += (memoryEnd-memoryStart);

  (void)count;
  (void)samples;
  (void)lines;
  return out;
}
