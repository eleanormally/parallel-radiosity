#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "fp.h"

f3 inline __device__ sub(f3 a, f3 b) {
  f3 out;
  out.data[0] = a.data[0]-b.data[0];
  out.data[1] = a.data[1]-b.data[1];
  out.data[2] = a.data[2]-b.data[2];
  return out;
}
double inline __device__ dot(f3 a, f3 b) {
  double out = 0;
  out += a.data[0]*b.data[0];
  out += a.data[1]*b.data[1];
  out += a.data[2]*b.data[2];
  return out;
}
double inline __device__ len(f3 a) {
  double len = 0;
  len += a.data[0]*a.data[0];
  len += a.data[1]*a.data[1];
  len += a.data[2]*a.data[2];
  return sqrt(len);
}



void __global__ calculate(FP** lines, double* out) {
  extern __shared__ double lineFactor[];
  FP p = lines[blockIdx.x][threadIdx.x];
  f3 iToJ = sub(p.a,p.b);
  f3 jToI = sub(p.b,p.a);
  double iCos = dot(p.anorm, iToJ) / len(iToJ);
  double jCos = dot(p.bnorm, jToI) / len(jToI);
  lineFactor[threadIdx.x] = fabs(iCos*jCos)/ (len(iToJ)*len(jToI));
  __syncthreads();
  for(unsigned int s=blockDim.x/2; s > 0; s >>= 1) {
    if(threadIdx.x < s) {
      lineFactor[threadIdx.x] += lineFactor[threadIdx.x+s];
    }
    __syncthreads();
  }
  if(threadIdx.x == 0) {
    out[blockIdx.x] = lineFactor[0];
  }
}

double* getFormFactor(FP** lines, int count, int samples, int rank) {
  int c;
  hipGetDeviceCount(&c);
  hipSetDevice(rank % c);
  double* cudaOut;
  FP** cudaLines;
  hipMallocManaged(&cudaLines, count*sizeof(FP*));
  for(int i = 0; i < count; i++) {
    hipMallocManaged(cudaLines+i,samples*sizeof(FP));
    for(int j = 0; j < samples; j++) {
      cudaLines[i][j] = lines[i][j];
    }
  }
  hipMallocManaged(&cudaOut, count*sizeof(double));
  calculate<<<count, samples, samples*sizeof(double)>>>(cudaLines, cudaOut);
  hipDeviceSynchronize();
  double* out = (double*)calloc(count, sizeof(double));
  for(int i = 0; i < count; i++) {
    out[i] = cudaOut[i];
    hipFree(cudaLines[i]);
  }
  hipFree(cudaLines);
  hipFree(cudaOut);

  (void)count;
  (void)samples;
  (void)lines;
  return out;
}
